#include "hip/hip_runtime.h"
/*
See repkv.cu for details. This is the backward pass of repkv forward.
Block size 128 seems fastest on H100
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "common.h"

// cpu reference code
void repkv_backward_cpu(float* dinp, const float* dout,
                       const int B, const int T, const int Cout,
                       const int hd, const int qh, const int kh, const int vh) {

    assert(Cout == (hd * (3 * qh)));
    assert(kh == vh);
    int nrep = qh / kh; // number of times to replicate key/value vectors
    int Cin = hd * (qh + kh + vh); // output channels

    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            // seek to the input position dout[b,t,:]
            const float* x = dout + b * T * Cout + t * Cout;
            // seek to the output position out[b,t,:]
            float* y = dinp + b * T * Cin + t * Cin;
            // copy all the query vectors, no changes
            for (int i = 0; i < hd * qh; i++) { y[i] = x[i]; }
            x += hd * qh; // advance input pointer
            y += hd * qh; // advance output pointer
            // copy key vectors, and replicate them nrep times
            for (int h = 0; h < kh; h++) {
                for (int n = 0; n < nrep; n++) {
                    for (int i = 0; i < hd; i++) { y[i] += x[i]; }
                    x += hd; // advance input pointer
                }
                y += hd; // advance output pointer
            }
            // copy value vectors, and replicate them nrep times
            for (int h = 0; h < vh; h++) {
                for (int n = 0; n < nrep; n++) {
                    for (int i = 0; i < hd; i++) { y[i] += x[i]; }
                    x += hd; // advance input pointer
                }
                y += hd; // advance output pointer
            }
        }
    }
}

// kernels
__global__ void repkv_backward_kernel2(floatX* dinp, const floatX* dout,
                                int B, int N, int NH, int replicate_factor, int HD) {
    // we have a single tensor dout of shapae of (B, N 3 * NH * HD)
    // we want to reduce sum (for K and V) into  (B, N, (NH + 2*(NH/replicate_factor)) * HD)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= B * N * 3 * NH * HD) { return;}
    int dinp_idx = idx; // keep backup

    int NKV = NH / replicate_factor;
    int nkv_factor = (replicate_factor + 2);   // replicate_factor is for (replicate_factor * NKV == NQ), 2 for K V

    // decode the dinp index
    int d = idx % HD;
    idx /= HD;
    int nkv = idx % NKV;
    idx /= NKV;
    int c = idx % nkv_factor;
    idx /= nkv_factor;
    int n = idx % N;
    int b = idx / N;

    int dout_idx;
    int nh_total = 3 * NH;
    // int nh_total = NH + 2 * (NH / replicate_factor);

    if (c >= 0 && c < nkv_factor - 2) {
        dout_idx = b * N * nh_total * HD + n * nh_total * HD + c * NKV * HD + nkv * HD + d;
        dinp[dinp_idx] = __ldcs(&dout[dout_idx]);
    // } else if (c == nkv_factor - 2) {
    //     // if (nkv % replicate_factor == 0) {
    //         float reduced_sum = 0;
    //         dout_idx = b * N * nh_total * HD + n * nh_total * HD + c * NKV * HD + nkv * HD + d;
    //         for (int i = 0; i < replicate_factor; i++) {
    //             reduced_sum += __ldcs(&dout[dout_idx+HD*i]);
    //         }
    //         dinp[dinp_idx] = reduced_sum;
    //     // }

    } else {
        // if (nkv % replicate_factor == 0) {
            float reduced_sum = 0;
            dout_idx = b * N * nh_total * HD + n * nh_total * HD + c * NKV * HD + nkv * HD + d;
            for (int i = 0; i < replicate_factor; i++) {
                reduced_sum += __ldcs(&dout[dout_idx+HD*i]);
            }
            dinp[dinp_idx] = reduced_sum;

        // }
    }
}

// kernels
__global__ void repkv_backward_kernel1(floatX* dinp, const floatX* dout,
                                int B, int N, int NH, int replicate_factor, int HD) {
    // we have a single tensor dout of shapae of (B, N 3 * NH * HD)
    // we want to reduce sum (for K and V) into  (B, N, (NH + 2*(NH/replicate_factor)) * HD)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= B * N * 3 * NH * HD) { return;}
    int dout_idx = idx; // keep backup

    // decode the dout index
    int d = idx % HD;
    idx /= HD;
    int nh = idx % NH;
    idx /= NH;
    int c = idx % 3;
    idx /= 3;
    int n = idx % N;
    int b = idx / N;

    int dinp_idx;
    int nh_total = NH + 2 * (NH / replicate_factor);

    if (c == 0) {
        dinp_idx = b * N * nh_total * HD + n * nh_total * HD + 0 * NH * HD + nh * HD + d;
        dinp[dinp_idx] = __ldcs(&dout[dout_idx]);
    } else if (c == 1) {
        if (nh % replicate_factor == 0) {
            float reduced_sum = 0;
            for (int i = 0; i < replicate_factor; i++) {
                reduced_sum += __ldcs(&dout[dout_idx+HD*i]);
            }

            dinp_idx = b * N * nh_total * HD + n * nh_total * HD + 1 * NH * HD + (nh / replicate_factor) * HD + d;
            dinp[dinp_idx] = reduced_sum;
        }

    } else {
        if (nh % replicate_factor == 0) {
            float reduced_sum = 0;
            for (int i = 0; i < replicate_factor; i++) {
                reduced_sum += __ldcs(&dout[dout_idx+HD*i]);
            }
            dinp_idx = b * N * nh_total * HD + n * nh_total * HD + (NH * HD + (NH / replicate_factor) * HD) + (nh / replicate_factor) * HD + d;
            dinp[dinp_idx] = reduced_sum;
        }
    }
}

// kernel launchers
void repkv_backward2(floatX* dinp, const floatX* dout,
    const int B, const int T, const int NH, const int NH_KV, const int d, int block_size) {
    int total_threads = B * T * (NH + 2 * NH_KV) * d;
    int num_blocks = ceil_div(total_threads, block_size);
    int replicate_factor = NH / NH_KV;
    repkv_backward_kernel2<<<num_blocks, block_size>>>(dinp, dout, B, T, NH, replicate_factor, d);
    cudaCheck(hipGetLastError());
}

// kernel launchers
void repkv_backward1(floatX* dinp, const floatX* dout,
    const int B, const int T, const int NH, const int NH_KV, const int d, int block_size) {
    int total_threads = B * T * (3 * NH) * d;
    int num_blocks = ceil_div(total_threads, block_size);
    int replicate_factor = NH / NH_KV;
    repkv_backward_kernel1<<<num_blocks, block_size>>>(dinp, dout, B, T, NH, replicate_factor, d);
    cudaCheck(hipGetLastError());
}

// kernel dispatcher
void repkv_backward(int kernel_num,
                   floatX* dinp, const floatX* dout,
                   int B, int T, int NH, int NH_KV, int d,
                   int block_size) {
    switch (kernel_num) {
        case 1:
            repkv_backward1(dinp, dout, B, T, NH, NH_KV, d, block_size);
            break;
        case 2:
            repkv_backward2(dinp, dout, B, T, NH, NH_KV, d, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}
#ifdef DEBUG
static void log_mat(float *inp, int B, int T, int C, int hd, int qh, int kh, int vh, char *title)
{
    printf("%s -----\n", title);
    for (int b = 0; b < B; b++) {
        printf("batch : %d ", b);
        for (int t = 0; t < T; t++) {
            printf("t = %d\n", t);
            const float* x = inp + b * T * C + t * C;
            printf("Query\n");
            for (int h=0; h < qh; h++) {
                for (int i = 0; i < hd; i++) {
                    printf("%f ", x[i]);
                }
                x += hd; // advance input pointer
                printf("\n");
            }
            printf("Key\n");
            for (int h=0; h < kh; h++) {
                for (int i = 0; i < hd; i++) {
                    printf("%f ", x[i]);
                }
                x += hd; // advance input pointer
                printf("\n");
            }
            printf("Value\n");
            for (int h=0; h < vh; h++) {
                for (int i = 0; i < hd; i++) {
                    printf("%f ", x[i]);
                }
                x += hd; // advance input pointer
                printf("\n");
            }
        }
    }
    printf("\n");
}
#endif // DEBUG

// tester
int main(int argc, char **argv) {
    srand(0);
#ifdef DEBUG
    int B = 1;
    int T = 2;
    int hd = 2; // head dim
    int qh = 4; // num query heads
    int kh = 2; // num key heads
    int vh = 2; // num value heads
#else
    int B = 8;
    int T = 1024;
    int hd = 128; // head dim
    int qh = 32; // num query heads
    int kh = 8; // num key heads
    int vh = 8; // num value heads
#endif

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    int Cout = hd * (qh * 3); // out, upstream channels
    int Cin = hd * (qh + kh + vh); // in, downstream channels

    // allocate (and fill) CPU memory
    float* dinp = (float*)malloc(B * T * Cin * sizeof(float));
    memset(dinp, 0, B * T * Cin * sizeof(float));
    float* dout = make_random_float(B * T * Cout * sizeof(float));

    // allocate GPU memory
    float* d_dinp;
    float* d_inp;
    float* d_dout;
    cudaCheck(hipMalloc(&d_dinp, B * T * Cin * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * Cin * sizeof(float)));
    cudaCheck(hipMalloc(&d_dout, B * T * Cout * sizeof(float)));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

#ifdef DEBUG
    int nrep = qh/kh;
    log_mat(dout, B, T, Cout, hd, qh, nrep*kh, nrep*vh, "dout");
#endif // DEBUG

    // CPU reference calculate
    repkv_backward_cpu(dinp, dout, B, T, Cout, hd, qh, kh, vh);

#ifdef DEBUG
    log_mat(dinp, B, T, Cin, hd, qh, kh, vh, "dinp");
#endif // DEBUG

    // check the correctness of the kernel at all block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    cudaCheck(hipMemcpy(d_dout, dout, B * T * Cout * sizeof(float), hipMemcpyHostToDevice));
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        repkv_backward(kernel_num, d_dinp, d_dout, B, T, qh, kh, hd, block_size);
        validate_result(d_dinp, dinp, "out", B * T * Cin, 1e-5f);
    }
    printf("All results match. Starting benchmarks.\n\n");

    // now benchmark
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        int repeat_times = 1000;
        float elapsed_time = benchmark_kernel(repeat_times, repkv_backward, kernel_num,
                                            d_dinp, d_dout, B, T, qh, kh, hd, block_size);
        printf("block_size %4d time %.4f ms\n", block_size, elapsed_time);
    }

    // free memory
    free(dinp);
    free(dout);
    cudaCheck(hipFree(d_dinp));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_dout));

    return 0;
}

